#include "hip/hip_runtime.h"
//
// cutrace_wavefront kernel and device function implementation.
//

#include <hip/hip_runtime.h>

#include "yocto_color.h"
#include "yocto_geometry.h"
#include "yocto_math.h"
#include "yocto_sampling.h"
#include "yocto_shading.h"

// HACK TO ALLOW CUT&PASTING FROM YOCTO'S CODE
#define inline __forceinline__ __device__
#define static static __forceinline__ __device__
#define optix_shader extern "C" __global__
#define optix_constant extern "C" __constant__

// whether to use builtin compound types or yocto's ones
#define CUTRACE_BUILTIN_VECS 0

// -----------------------------------------------------------------------------
// SUBSTITUTES FOR STD TYPES
// -----------------------------------------------------------------------------
namespace yocto {

// pair
template <typename T1, typename T2>
struct pair_ {
  T1 first;
  T2 second;
};

template <typename T>
struct span {
  inline bool   empty() const { return _size == 0; }
  inline size_t size() const { return _size; }

  inline T&       operator[](int idx) { return _data[idx]; }
  inline const T& operator[](int idx) const { return _data[idx]; }
  inline T&       at(int idx) { return _data[idx]; }
  inline const T& at(int idx) const { return _data[idx]; }

  inline T*       begin() { return _data; }
  inline T*       end() { return _data + _size; }
  inline const T* begin() const { return _data; }
  inline const T* end() const { return _data + _size; }

  inline T&       front() { return *_data; }
  inline T&       back() { return *(_data + _size - 1); }
  inline const T& front() const { return *_data; }
  inline const T& back() const { return *(_data + _size - 1); }

  inline T*       data() { return _data; }
  inline const T* data() const { return _data; }

  T*     _data = nullptr;
  size_t _size = 0;
};

}  // namespace yocto

// -----------------------------------------------------------------------------
// SAMPLING FUNCTIONS
// -----------------------------------------------------------------------------
namespace yocto {

// simplified version of possible implementation from cpprenference.com
template <class T>
static const T* _upper_bound(const T* first, const T* last, const T& value) {
  const T*  it;
  ptrdiff_t count, step;
  count = last - first;

  while (count > 0) {
    it   = first;
    step = count / 2;
    it += step;
    if (!(value < *it)) {
      first = ++it;
      count -= step + 1;
    } else
      count = step;
  }
  return first;
}

// Sample a discrete distribution represented by its cdf.
inline int sample_discrete(const span<float>& cdf, float r) {
  r = clamp(r * cdf.back(), (float)0, cdf.back() - (float)0.00001);
  auto idx =
      (int)(_upper_bound(cdf.data(), cdf.data() + cdf.size(), r) - cdf.data());
  return clamp(idx, 0, (int)cdf.size() - 1);
}
// Pdf for uniform discrete distribution sampling.
inline float sample_discrete_pdf(const span<float>& cdf, int idx) {
  if (idx == 0) return cdf.at(0);
  return cdf.at(idx) - cdf.at(idx - 1);
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// CUDA HELPERS
// -----------------------------------------------------------------------------
namespace yocto {

template <typename T>
struct cuspan {
  inline bool     empty() const { return _size == 0; }
  inline size_t   size() const { return _size; }
  inline T&       operator[](int idx) { return _data[idx]; }
  inline const T& operator[](int idx) const { return _data[idx]; }

  inline T*       begin() { return _data; }
  inline T*       end() { return _data + _size; }
  inline const T* begin() const { return _data; }
  inline const T* end() const { return _data + _size; }

  inline T&       front() { return *_data; }
  inline T&       back() { return *(_data + _size - 1); }
  inline const T& front() const { return *_data; }
  inline const T& back() const { return *(_data + _size - 1); }

  inline operator span<T>() const { return {_data, _size}; }

  T*     _data = nullptr;
  size_t _size = 0;
};

template <typename T, size_t Size = 16>
struct svector {
  inline bool     empty() const { return _size == 0; }
  inline size_t   size() const { return _size; }
  inline T&       operator[](int idx) { return _data[idx]; }
  inline const T& operator[](int idx) const { return _data[idx]; }

  inline T*       begin() { return _data; }
  inline T*       end() { return _data + _size; }
  inline const T* begin() const { return _data; }
  inline const T* end() const { return _data + _size; }

  inline T&       front() { return *_data; }
  inline T&       back() { return *(_data + _size - 1); }
  inline const T& front() const { return *_data; }
  inline const T& back() const { return *(_data + _size - 1); }

  inline void push_back(const T& value) { _data[_size++] = value; }
  inline void pop_back() { _size--; }

  T      _data[Size] = {};
  size_t _size       = 0;
};

inline void* unpackPointer(uint32_t i0, uint32_t i1) {
  const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
  void*          ptr  = reinterpret_cast<void*>(uptr);
  return ptr;
}

inline void packPointer(void* ptr, uint32_t& i0, uint32_t& i1) {
  const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
  i0                  = uptr >> 32;
  i1                  = uptr & 0x00000000ffffffff;
}

template <typename T>
inline T* getPRD() {
  const uint32_t u0 = optixGetPayload_0();
  const uint32_t u1 = optixGetPayload_1();
  return reinterpret_cast<T*>(unpackPointer(u0, u1));
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// CUTRACE TYPES
// -----------------------------------------------------------------------------
namespace yocto {

constexpr int invalidid = -1;
struct material_point;

struct cutrace_path {
  cuspan<vec3f>          radiance      = {};
  cuspan<vec3f>          weights       = {};
  cuspan<ray3f>          rays          = {};
  cuspan<material_point> volume_back   = {};
  cuspan<bool>           volume_empty  = {};
  cuspan<float>          max_roughness = {};
  cuspan<bool>           hit           = {};
  cuspan<vec3f>          hit_albedo    = {};
  cuspan<vec3f>          hit_normal    = {};
  cuspan<int>            opbounces     = {};
  cuspan<int>            bounces       = {};
};

struct cutrace_intersection {
  cuspan<int>   instance = {};
  cuspan<int>   element  = {};
  cuspan<vec2f> uv       = {};
  cuspan<float> distance = {};
  cuspan<bool>  hit      = {};
};

struct cutrace_state {
  int               width            = 0;
  int               height           = 0;
  int               samples          = 0;
  cuspan<vec4f>     image            = {};
  cuspan<vec3f>     albedo           = {};
  cuspan<vec3f>     normal           = {};
  cuspan<int>       pixel_samples    = {};
  cuspan<rng_state> rngs             = {};
  cuspan<vec4f>     denoised         = {};
  cuspan<byte>      denoiser_state   = {};
  cuspan<byte>      denoiser_scratch = {};

  cutrace_path         path         = {};
  cutrace_intersection intersection = {};
};

struct cucamera_data {
  frame3f frame        = {};
  float   lens         = {};
  float   film         = {};
  float   aspect       = {};
  float   focus        = {};
  float   aperture     = {};
  bool    orthographic = {};
};

struct cutexture_data {
  int                 width   = 0;
  int                 height  = 0;
  bool                linear  = false;
  bool                nearest = false;
  bool                clamp   = false;
  hipTextureObject_t texture = 0;
  hipArray_t         array   = nullptr;
};

enum struct material_type {
  // clang-format off
  matte, glossy, reflective, transparent, refractive, subsurface, volumetric, 
  gltfpbr
  // clang-format on
};

struct cumaterial_data {
  material_type type         = material_type::matte;
  vec3f         emission     = {0, 0, 0};
  vec3f         color        = {0, 0, 0};
  float         roughness    = 0;
  float         metallic     = 0;
  float         ior          = 1.5f;
  vec3f         scattering   = {0, 0, 0};
  float         scanisotropy = 0;
  float         trdepth      = 0.01f;
  float         opacity      = 1;

  int emission_tex   = invalidid;
  int color_tex      = invalidid;
  int roughness_tex  = invalidid;
  int scattering_tex = invalidid;
  int normal_tex     = invalidid;
};

struct cuinstance_data {
  frame3f frame    = {{1, 0, 0}, {0, 1, 0}, {0, 0, 1}, {0, 0, 0}};
  int     shape    = -1;
  int     material = -1;
};

struct cushape_data {
  cuspan<vec3f> positions = {};
  cuspan<vec3f> normals   = {};
  cuspan<vec2f> texcoords = {};
  cuspan<vec4f> colors    = {};
  cuspan<vec3i> triangles = {};
};

struct cuenvironment_data {
  frame3f frame        = {{1, 0, 0}, {0, 1, 0}, {0, 0, 1}, {0, 0, 0}};
  vec3f   emission     = {0, 0, 0};
  int     emission_tex = invalidid;
};

struct cuscene_data {
  cuspan<cucamera_data>      cameras      = {};
  cuspan<cutexture_data>     textures     = {};
  cuspan<cumaterial_data>    materials    = {};
  cuspan<cushape_data>       shapes       = {};
  cuspan<cuinstance_data>    instances    = {};
  cuspan<cuenvironment_data> environments = {};
};

struct bvh_node {
  bbox3f  bbox     = invalidb3f;
  int32_t start    = 0;
  int16_t num      = 0;
  int8_t  axis     = 0;
  bool    internal = false;
};

struct cubvh_tree {
  cuspan<bvh_node> nodes      = {};
  cuspan<int>      primitives = {};
};

struct cushape_bvh {
  cubvh_tree bvh = {};
};

struct cuscene_bvh {
  cubvh_tree          bvh    = {};
  cuspan<cushape_bvh> shapes = {};
};

// Type of tracing algorithm
enum struct trace_sampler_type {
  path,        // path tracing
  pathdirect,  // path tracing with direct
  pathmis,     // path tracing with mis
  pathtest,    // path tracing for testing
  naive,       // naive path tracing
  eyelight,    // eyelight rendering
  diagram,     // diagram rendering
  furnace,     // furnace test
  falsecolor,  // false color rendering
};
// Type of false color visualization
enum struct trace_falsecolor_type {
  // clang-format off
  position, normal, frontfacing, gnormal, gfrontfacing, texcoord, mtype, color,
  emission, roughness, opacity, metallic, delta, instance, shape, material, 
  element, highlight
  // clang-format on
};

// Default trace seed
constexpr auto trace_default_seed = 961748941ull;

// params
struct trace_params {
  int                   camera         = 0;
  int                   resolution     = 1280;
  trace_sampler_type    sampler        = trace_sampler_type::path;
  trace_falsecolor_type falsecolor     = trace_falsecolor_type::color;
  int                   samples        = 512;
  int                   bounces        = 8;
  float                 clamp          = 10;
  bool                  nocaustics     = false;
  bool                  envhidden      = false;
  bool                  tentfilter     = false;
  uint64_t              seed           = trace_default_seed;
  bool                  embreebvh      = false;
  bool                  highqualitybvh = false;
  bool                  noparallel     = false;
  int                   pratio         = 8;
  bool                  denoise        = false;
  int                   batch          = 1;
};

using cutrace_bvh = cuscene_bvh;

// light
struct cutrace_light {
  int           instance     = invalidid;
  int           environment  = invalidid;
  cuspan<float> elements_cdf = {};
};

// lights
struct cutrace_lights {
  cuspan<cutrace_light> lights = {};
};

struct cutrace_globals {
  cutrace_state  state  = {};
  cuscene_data   scene  = {};
  cuscene_bvh    bvh    = {};
  cutrace_lights lights = {};
  trace_params   params = {};
};

// global data
__constant__ cutrace_globals globals;

// compatibility aliases
using trace_bvh    = cutrace_bvh;
using trace_lights = cutrace_lights;

}  // namespace yocto

// -----------------------------------------------------------------------------
// SCENE FUNCTIONS
// -----------------------------------------------------------------------------
namespace yocto {

// compatibility aliases
using scene_data       = cuscene_data;
using camera_data      = cucamera_data;
using material_data    = cumaterial_data;
using texture_data     = cutexture_data;
using instance_data    = cuinstance_data;
using shape_data       = cushape_data;
using environment_data = cuenvironment_data;

// constant values
constexpr auto min_roughness = 0.03f * 0.03f;

// Evaluates an image at a point `uv`.
static vec4f eval_texture(const texture_data& texture, const vec2f& texcoord,
    bool as_linear = false, bool no_interpolation = false,
    bool clamp_to_edge = false) {
  auto fromTexture = tex2D<float4>(texture.texture, texcoord.x, texcoord.y);
  auto color       = vec4f{
      fromTexture.x, fromTexture.y, fromTexture.z, fromTexture.w};
  if (as_linear && !texture.linear) {
    return srgb_to_rgb(color);
  } else {
    return color;
  }
}

// Helpers
static vec4f eval_texture(const scene_data& scene, int texture, const vec2f& uv,
    bool ldr_as_linear = false, bool no_interpolation = false,
    bool clamp_to_edge = false) {
  if (texture == invalidid) return {1, 1, 1, 1};
  return eval_texture(
      scene.textures[texture], uv, ldr_as_linear, no_interpolation);
}

// Material parameters evaluated at a point on the surface
struct material_point {
  material_type type         = material_type::gltfpbr;
  vec3f         emission     = {0, 0, 0};
  vec3f         color        = {0, 0, 0};
  float         opacity      = 1;
  float         roughness    = 0;
  float         metallic     = 0;
  float         ior          = 1;
  vec3f         density      = {0, 0, 0};
  vec3f         scattering   = {0, 0, 0};
  float         scanisotropy = 0;
  float         trdepth      = 0.01f;
};

// Eval position
static vec3f eval_position(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv) {
  auto& shape = scene.shapes[instance.shape];
  if (!shape.triangles.empty()) {
    auto t = shape.triangles[element];
    return transform_point(
        instance.frame, interpolate_triangle(shape.positions[t.x],
                            shape.positions[t.y], shape.positions[t.z], uv));
  } else {
    return {0, 0, 0};
  }
}

// Shape element normal.
static vec3f eval_element_normal(
    const scene_data& scene, const instance_data& instance, int element) {
  auto& shape = scene.shapes[instance.shape];
  if (!shape.triangles.empty()) {
    auto t = shape.triangles[element];
    return transform_normal(
        instance.frame, triangle_normal(shape.positions[t.x],
                            shape.positions[t.y], shape.positions[t.z]));
  } else {
    return {0, 0, 0};
  }
}

// Eval normal
static vec3f eval_normal(const scene_data& scene, const instance_data& instance,
    int element, const vec2f& uv) {
  auto& shape = scene.shapes[instance.shape];
  if (shape.normals.empty())
    return eval_element_normal(scene, instance, element);
  if (!shape.triangles.empty()) {
    auto t = shape.triangles[element];
    return transform_normal(
        instance.frame, normalize(interpolate_triangle(shape.normals[t.x],
                            shape.normals[t.y], shape.normals[t.z], uv)));
  } else {
    return {0, 0, 0};
  }
}

// Eval texcoord
static vec2f eval_texcoord(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv) {
  auto& shape = scene.shapes[instance.shape];
  if (shape.texcoords.empty()) return uv;
  if (!shape.triangles.empty()) {
    auto t = shape.triangles[element];
    return interpolate_triangle(
        shape.texcoords[t.x], shape.texcoords[t.y], shape.texcoords[t.z], uv);
  } else {
    return {0, 0};
  }
}

// Shape element normal.
static pair_<vec3f, vec3f> eval_element_tangents(
    const scene_data& scene, const instance_data& instance, int element) {
  auto& shape = scene.shapes[instance.shape];
  if (!shape.triangles.empty() && !shape.texcoords.empty()) {
    auto t   = shape.triangles[element];
    auto tuv = triangle_tangents_fromuv(shape.positions[t.x],
        shape.positions[t.y], shape.positions[t.z], shape.texcoords[t.x],
        shape.texcoords[t.y], shape.texcoords[t.z]);
    return {transform_direction(instance.frame, tuv.first),
        transform_direction(instance.frame, tuv.second)};
  } else {
    return {};
  }
}

static vec3f eval_normalmap(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv) {
  auto& shape    = scene.shapes[instance.shape];
  auto& material = scene.materials[instance.material];
  // apply normal mapping
  auto normal   = eval_normal(scene, instance, element, uv);
  auto texcoord = eval_texcoord(scene, instance, element, uv);
  if (material.normal_tex != invalidid && (!shape.triangles.empty())) {
    auto& normal_tex = scene.textures[material.normal_tex];
    auto  normalmap  = -1 + 2 * xyz(eval_texture(normal_tex, texcoord, false));
    auto  tuv        = eval_element_tangents(scene, instance, element);
    auto  frame      = frame3f{tuv.first, tuv.second, normal, {0, 0, 0}};
    frame.x          = orthonormalize(frame.x, frame.z);
    frame.y          = normalize(cross(frame.z, frame.x));
    auto flip_v      = dot(frame.y, tuv.second) < 0;
    normalmap.y *= flip_v ? 1 : -1;  // flip vertical axis
    normal = transform_normal(frame, normalmap);
  }
  return normal;
}

// Eval shading position
static vec3f eval_shading_position(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv,
    const vec3f& outgoing) {
  auto& shape = scene.shapes[instance.shape];
  if (!shape.triangles.empty()) {
    return eval_position(scene, instance, element, uv);
  } else {
    return {0, 0, 0};
  }
}

// Eval shading normal
static vec3f eval_shading_normal(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv,
    const vec3f& outgoing) {
  auto& shape    = scene.shapes[instance.shape];
  auto& material = scene.materials[instance.material];
  if (!shape.triangles.empty()) {
    auto normal = eval_normal(scene, instance, element, uv);
    if (material.normal_tex != invalidid) {
      normal = eval_normalmap(scene, instance, element, uv);
    }
    if (material.type == material_type::refractive) return normal;
    return dot(normal, outgoing) >= 0 ? normal : -normal;
  } else {
    return {0, 0, 0};
  }
}

// Eval color
static vec4f eval_color(const scene_data& scene, const instance_data& instance,
    int element, const vec2f& uv) {
  auto& shape = scene.shapes[instance.shape];
  if (shape.colors.empty()) return {1, 1, 1, 1};
  if (!shape.triangles.empty()) {
    auto t = shape.triangles[element];
    return interpolate_triangle(
        shape.colors[t.x], shape.colors[t.y], shape.colors[t.z], uv);
  } else {
    return {0, 0, 0, 0};
  }
}

// Evaluate material
static material_point eval_material(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv) {
  auto& material = scene.materials[instance.material];
  auto  texcoord = eval_texcoord(scene, instance, element, uv);

  // evaluate textures
  auto emission_tex = eval_texture(
      scene, material.emission_tex, texcoord, true);
  auto color_shp     = eval_color(scene, instance, element, uv);
  auto color_tex     = eval_texture(scene, material.color_tex, texcoord, true);
  auto roughness_tex = eval_texture(
      scene, material.roughness_tex, texcoord, false);
  auto scattering_tex = eval_texture(
      scene, material.scattering_tex, texcoord, true);

  // material point
  auto point         = material_point{};
  point.type         = material.type;
  point.emission     = material.emission * xyz(emission_tex);
  point.color        = material.color * xyz(color_tex) * xyz(color_shp);
  point.opacity      = material.opacity * color_tex.w * color_shp.w;
  point.metallic     = material.metallic * roughness_tex.z;
  point.roughness    = material.roughness * roughness_tex.y;
  point.roughness    = point.roughness * point.roughness;
  point.ior          = material.ior;
  point.scattering   = material.scattering * xyz(scattering_tex);
  point.scanisotropy = material.scanisotropy;
  point.trdepth      = material.trdepth;

  // volume density
  if (material.type == material_type::refractive ||
      material.type == material_type::volumetric ||
      material.type == material_type::subsurface) {
    point.density = -log(clamp(point.color, 0.0001f, 1.0f)) / point.trdepth;
  } else {
    point.density = {0, 0, 0};
  }

  // fix roughness
  if (point.type == material_type::matte ||
      point.type == material_type::gltfpbr ||
      point.type == material_type::glossy) {
    point.roughness = clamp(point.roughness, min_roughness, 1.0f);
  } else if (material.type == material_type::volumetric) {
    point.roughness = 0;
  } else {
    if (point.roughness < min_roughness) point.roughness = 0;
  }

  return point;
}

static bool is_volumetric(const material_data& material) {
  return material.type == material_type::refractive ||
         material.type == material_type::volumetric ||
         material.type == material_type::subsurface;
}

// check if an instance is volumetric
static bool is_volumetric(
    const scene_data& scene, const instance_data& instance) {
  return is_volumetric(scene.materials[instance.material]);
}

// check if a brdf is a delta
static bool is_delta(const material_point& material) {
  return (material.type == material_type::reflective &&
             material.roughness == 0) ||
         (material.type == material_type::refractive &&
             material.roughness == 0) ||
         (material.type == material_type::transparent &&
             material.roughness == 0) ||
         (material.type == material_type::volumetric);
}

static ray3f eval_camera(
    const cucamera_data& camera, const vec2f& image_uv, const vec2f& lens_uv) {
  auto film = camera.aspect >= 1
                  ? vec2f{camera.film, camera.film / camera.aspect}
                  : vec2f{camera.film * camera.aspect, camera.film};
  auto q    = vec3f{
      film.x * (0.5f - image_uv.x), film.y * (image_uv.y - 0.5f), camera.lens};
  // ray direction through the lens center
  auto dc = -normalize(q);
  // point on the lens
  auto e = vec3f{
      lens_uv.x * camera.aperture / 2, lens_uv.y * camera.aperture / 2, 0};
  // point on the focus plane
  auto p = dc * camera.focus / abs(dc.z);
  // correct ray direction to account for camera focusing
  auto d = normalize(p - e);
  // done
  return ray3f{
      transform_point(camera.frame, e), transform_direction(camera.frame, d)};
}

// Evaluate environment color.
static vec3f eval_environment(const scene_data& scene,
    const environment_data& environment, const vec3f& direction) {
  auto wl       = transform_direction_inverse(environment.frame, direction);
  auto texcoord = vec2f{
      atan2(wl.z, wl.x) / (2 * pif), acos(clamp(wl.y, -1.0f, 1.0f)) / pif};
  if (texcoord.x < 0) texcoord.x += 1;
  return environment.emission *
         xyz(eval_texture(scene, environment.emission_tex, texcoord));
}

// Evaluate all environment color.
static vec3f eval_environment(const scene_data& scene, const vec3f& direction) {
  auto emission = vec3f{0, 0, 0};
  for (auto& environment : scene.environments) {
    emission += eval_environment(scene, environment, direction);
  }
  return emission;
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// RAY-SCENE INTERSECTION
// -----------------------------------------------------------------------------
namespace yocto {

// intersection result
struct scene_intersection {
  int   instance = -1;
  int   element  = -1;
  vec2f uv       = {0, 0};
  float distance = 0;
  bool  hit      = false;
  float _pad     = 0;
};

struct shape_intersection {
  int   element  = -1;
  vec2f uv       = {0, 0};
  float distance = 0;
  bool  hit      = false;
};

static shape_intersection intersect_shape_bvh(
    const cushape_bvh& sbvh, const shape_data& shape, const ray3f& ray_) {
  // get bvh tree
  auto& bvh = sbvh.bvh;

  // check empty
  if (bvh.nodes.empty()) return {};

  // node stack
  int  node_stack[128];
  auto node_cur          = 0;
  node_stack[node_cur++] = 0;

  // shared variables
  auto intersection = shape_intersection{};

  // copy ray to modify it
  auto ray = ray_;

  // prepare ray for fast queries
  auto ray_dinv  = vec3f{1 / ray.d.x, 1 / ray.d.y, 1 / ray.d.z};
  auto ray_dsign = vec3i{(ray_dinv.x < 0) ? 1 : 0, (ray_dinv.y < 0) ? 1 : 0,
      (ray_dinv.z < 0) ? 1 : 0};

  // walking stack
  while (node_cur != 0) {
    // grab node
    auto& node = bvh.nodes[node_stack[--node_cur]];

    // intersect bbox
    // if (!intersect_bbox(ray, ray_dinv, ray_dsign, node.bbox)) continue;
    if (!intersect_bbox(ray, ray_dinv, node.bbox)) continue;

    // intersect node, switching based on node type
    // for each type, iterate over the the primitive list
    if (node.internal) {
      // for internal nodes, attempts to proceed along the
      // split axis from smallest to largest nodes
      if (ray_dsign[node.axis] != 0) {
        node_stack[node_cur++] = node.start + 0;
        node_stack[node_cur++] = node.start + 1;
      } else {
        node_stack[node_cur++] = node.start + 1;
        node_stack[node_cur++] = node.start + 0;
      }
    } else if (!shape.triangles.empty()) {
      for (auto idx = node.start; idx < node.start + node.num; idx++) {
        auto& t             = shape.triangles[bvh.primitives[idx]];
        auto  pintersection = intersect_triangle(ray, shape.positions[t.x],
             shape.positions[t.y], shape.positions[t.z]);
        if (!pintersection.hit) continue;
        intersection = {bvh.primitives[idx], pintersection.uv,
            pintersection.distance, true};
        ray.tmax     = pintersection.distance;
      }
    }
  }

  return intersection;
}

static scene_intersection intersect_scene(
    const cuscene_bvh& sbvh, const scene_data& scene, const ray3f& ray_) {
  // get instances bvh
  auto& bvh = sbvh.bvh;

  // check empty
  if (bvh.nodes.empty()) return {};

  // node stack
  int  node_stack[128];
  auto node_cur          = 0;
  node_stack[node_cur++] = 0;

  // intersection
  auto intersection = scene_intersection{};

  // copy ray to modify it
  auto ray = ray_;

  // prepare ray for fast queries
  auto ray_dinv  = vec3f{1 / ray.d.x, 1 / ray.d.y, 1 / ray.d.z};
  auto ray_dsign = vec3i{(ray_dinv.x < 0) ? 1 : 0, (ray_dinv.y < 0) ? 1 : 0,
      (ray_dinv.z < 0) ? 1 : 0};

  // walking stack
  while (node_cur != 0) {
    // grab node
    auto& node = bvh.nodes[node_stack[--node_cur]];

    // intersect bbox
    // if (!intersect_bbox(ray, ray_dinv, ray_dsign, node.bbox)) continue;
    if (!intersect_bbox(ray, ray_dinv, node.bbox)) continue;

    // intersect node, switching based on node type
    // for each type, iterate over the the primitive list
    if (node.internal) {
      // for internal nodes, attempts to proceed along the
      // split axis from smallest to largest nodes
      if (ray_dsign[node.axis] != 0) {
        node_stack[node_cur++] = node.start + 0;
        node_stack[node_cur++] = node.start + 1;
      } else {
        node_stack[node_cur++] = node.start + 1;
        node_stack[node_cur++] = node.start + 0;
      }
    } else {
      for (auto idx = node.start; idx < node.start + node.num; idx++) {
        auto& instance_ = scene.instances[bvh.primitives[idx]];
        auto  inv_ray   = transform_ray(inverse(instance_.frame, true), ray);
        auto  sintersection = intersect_shape_bvh(sbvh.shapes[instance_.shape],
             scene.shapes[instance_.shape], inv_ray);
        if (!sintersection.hit) continue;
        intersection = {bvh.primitives[idx], sintersection.element,
            sintersection.uv, sintersection.distance, true};
        ray.tmax     = sintersection.distance;
      }
    }
  }

  return intersection;
}

// instance intersection, for now manual
static scene_intersection intersect_instance(const trace_bvh& bvh,
    const cuscene_data& scene, int instance_id, const ray3f& ray) {
  auto& instance     = scene.instances[instance_id];
  auto& shape        = scene.shapes[instance.shape];
  auto  intersection = scene_intersection{};
  auto  tray         = ray3f{transform_point_inverse(instance.frame, ray.o),
      transform_vector_inverse(instance.frame, ray.d)};
  for (auto element = 0; element < shape.triangles.size(); element++) {
    auto& triangle = shape.triangles[element];
    auto  isec     = intersect_triangle(tray, shape.positions[triangle.x],
             shape.positions[triangle.y], shape.positions[triangle.z]);
    if (!isec.hit) continue;
    intersection.hit      = true;
    intersection.instance = instance_id;
    intersection.element  = element;
    intersection.uv       = isec.uv;
    intersection.distance = isec.distance;
    tray.tmax             = isec.distance;
  }
  return intersection;
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// TRACE FUNCTIONS
// -----------------------------------------------------------------------------
namespace yocto {

// Convenience functions
[[maybe_unused]] static vec3f eval_position(
    const scene_data& scene, const scene_intersection& intersection) {
  return eval_position(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv);
}
[[maybe_unused]] static vec3f eval_normal(
    const scene_data& scene, const scene_intersection& intersection) {
  return eval_normal(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv);
}
[[maybe_unused]] static vec3f eval_element_normal(
    const scene_data& scene, const scene_intersection& intersection) {
  return eval_element_normal(
      scene, scene.instances[intersection.instance], intersection.element);
}
[[maybe_unused]] static vec3f eval_shading_position(const scene_data& scene,
    const scene_intersection& intersection, const vec3f& outgoing) {
  return eval_shading_position(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv, outgoing);
}
[[maybe_unused]] static vec3f eval_shading_normal(const scene_data& scene,
    const scene_intersection& intersection, const vec3f& outgoing) {
  return eval_shading_normal(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv, outgoing);
}
[[maybe_unused]] static vec2f eval_texcoord(
    const scene_data& scene, const scene_intersection& intersection) {
  return eval_texcoord(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv);
}
[[maybe_unused]] static material_point eval_material(
    const scene_data& scene, const scene_intersection& intersection) {
  return eval_material(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv);
}
[[maybe_unused]] static bool is_volumetric(
    const scene_data& scene, const scene_intersection& intersection) {
  return is_volumetric(scene, scene.instances[intersection.instance]);
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// TRACE FUNCTIONS
// -----------------------------------------------------------------------------
namespace yocto {

// Evaluates/sample the BRDF scaled by the cosine of the incoming direction.
static vec3f eval_emission(const material_point& material, const vec3f& normal,
    const vec3f& outgoing) {
  return dot(normal, outgoing) >= 0 ? material.emission : vec3f{0, 0, 0};
}

// Evaluates/sample the BRDF scaled by the cosine of the incoming direction.
static vec3f eval_bsdfcos(const material_point& material, const vec3f& normal,
    const vec3f& outgoing, const vec3f& incoming) {
  if (material.roughness == 0) return {0, 0, 0};

  if (material.type == material_type::matte) {
    return eval_matte(material.color, normal, outgoing, incoming);
  } else if (material.type == material_type::glossy) {
    return eval_glossy(material.color, material.ior, material.roughness, normal,
        outgoing, incoming);
  } else if (material.type == material_type::reflective) {
    return eval_reflective(
        material.color, material.roughness, normal, outgoing, incoming);
  } else if (material.type == material_type::transparent) {
    return eval_transparent(material.color, material.ior, material.roughness,
        normal, outgoing, incoming);
  } else if (material.type == material_type::refractive) {
    return eval_refractive(material.color, material.ior, material.roughness,
        normal, outgoing, incoming);
  } else if (material.type == material_type::subsurface) {
    return eval_refractive(material.color, material.ior, material.roughness,
        normal, outgoing, incoming);
  } else if (material.type == material_type::gltfpbr) {
    return eval_gltfpbr(material.color, material.ior, material.roughness,
        material.metallic, normal, outgoing, incoming);
  } else {
    return {0, 0, 0};
  }
}

static vec3f eval_delta(const material_point& material, const vec3f& normal,
    const vec3f& outgoing, const vec3f& incoming) {
  if (material.roughness != 0) return {0, 0, 0};

  if (material.type == material_type::reflective) {
    return eval_reflective(material.color, normal, outgoing, incoming);
  } else if (material.type == material_type::transparent) {
    return eval_transparent(
        material.color, material.ior, normal, outgoing, incoming);
  } else if (material.type == material_type::refractive) {
    return eval_refractive(
        material.color, material.ior, normal, outgoing, incoming);
  } else if (material.type == material_type::volumetric) {
    return eval_passthrough(material.color, normal, outgoing, incoming);
  } else {
    return {0, 0, 0};
  }
}

// Picks a direction based on the BRDF
static vec3f sample_bsdfcos(const material_point& material, const vec3f& normal,
    const vec3f& outgoing, float rnl, const vec2f& rn) {
  if (material.roughness == 0) return {0, 0, 0};

  if (material.type == material_type::matte) {
    return sample_matte(material.color, normal, outgoing, rn);
  } else if (material.type == material_type::glossy) {
    return sample_glossy(material.color, material.ior, material.roughness,
        normal, outgoing, rnl, rn);
  } else if (material.type == material_type::reflective) {
    return sample_reflective(
        material.color, material.roughness, normal, outgoing, rn);
  } else if (material.type == material_type::transparent) {
    return sample_transparent(material.color, material.ior, material.roughness,
        normal, outgoing, rnl, rn);
  } else if (material.type == material_type::refractive) {
    return sample_refractive(material.color, material.ior, material.roughness,
        normal, outgoing, rnl, rn);
  } else if (material.type == material_type::subsurface) {
    return sample_refractive(material.color, material.ior, material.roughness,
        normal, outgoing, rnl, rn);
  } else if (material.type == material_type::gltfpbr) {
    return sample_gltfpbr(material.color, material.ior, material.roughness,
        material.metallic, normal, outgoing, rnl, rn);
  } else {
    return {0, 0, 0};
  }
}

static vec3f sample_delta(const material_point& material, const vec3f& normal,
    const vec3f& outgoing, float rnl) {
  if (material.roughness != 0) return {0, 0, 0};

  if (material.type == material_type::reflective) {
    return sample_reflective(material.color, normal, outgoing);
  } else if (material.type == material_type::transparent) {
    return sample_transparent(
        material.color, material.ior, normal, outgoing, rnl);
  } else if (material.type == material_type::refractive) {
    return sample_refractive(
        material.color, material.ior, normal, outgoing, rnl);
  } else if (material.type == material_type::volumetric) {
    return sample_passthrough(material.color, normal, outgoing);
  } else {
    return {0, 0, 0};
  }
}

// Compute the weight for sampling the BRDF
static float sample_bsdfcos_pdf(const material_point& material,
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  if (material.roughness == 0) return 0;

  if (material.type == material_type::matte) {
    return sample_matte_pdf(material.color, normal, outgoing, incoming);
  } else if (material.type == material_type::glossy) {
    return sample_glossy_pdf(material.color, material.ior, material.roughness,
        normal, outgoing, incoming);
  } else if (material.type == material_type::reflective) {
    return sample_reflective_pdf(
        material.color, material.roughness, normal, outgoing, incoming);
  } else if (material.type == material_type::transparent) {
    return sample_tranparent_pdf(material.color, material.ior,
        material.roughness, normal, outgoing, incoming);
  } else if (material.type == material_type::refractive) {
    return sample_refractive_pdf(material.color, material.ior,
        material.roughness, normal, outgoing, incoming);
  } else if (material.type == material_type::subsurface) {
    return sample_refractive_pdf(material.color, material.ior,
        material.roughness, normal, outgoing, incoming);
  } else if (material.type == material_type::gltfpbr) {
    return sample_gltfpbr_pdf(material.color, material.ior, material.roughness,
        material.metallic, normal, outgoing, incoming);
  } else {
    return 0;
  }
}

static float sample_delta_pdf(const material_point& material,
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  if (material.roughness != 0) return 0;

  if (material.type == material_type::reflective) {
    return sample_reflective_pdf(material.color, normal, outgoing, incoming);
  } else if (material.type == material_type::transparent) {
    return sample_tranparent_pdf(
        material.color, material.ior, normal, outgoing, incoming);
  } else if (material.type == material_type::refractive) {
    return sample_refractive_pdf(
        material.color, material.ior, normal, outgoing, incoming);
  } else if (material.type == material_type::volumetric) {
    return sample_passthrough_pdf(material.color, normal, outgoing, incoming);
  } else {
    return 0;
  }
}

static vec3f eval_scattering(const material_point& material,
    const vec3f& outgoing, const vec3f& incoming) {
  if (material.density == vec3f{0, 0, 0}) return {0, 0, 0};
  return material.scattering * material.density *
         eval_phasefunction(material.scanisotropy, outgoing, incoming);
}

static vec3f sample_scattering(const material_point& material,
    const vec3f& outgoing, float rnl, const vec2f& rn) {
  if (material.density == vec3f{0, 0, 0}) return {0, 0, 0};
  return sample_phasefunction(material.scanisotropy, outgoing, rn);
}

static float sample_scattering_pdf(const material_point& material,
    const vec3f& outgoing, const vec3f& incoming) {
  if (material.density == vec3f{0, 0, 0}) return 0;
  return sample_phasefunction_pdf(material.scanisotropy, outgoing, incoming);
}

// Sample camera
static ray3f sample_camera(const camera_data& camera, const vec2i& ij,
    const vec2i& image_size, const vec2f& puv, const vec2f& luv, bool tent) {
  if (!tent) {
    auto uv = vec2f{
        (ij.x + puv.x) / image_size.x, (ij.y + puv.y) / image_size.y};
    return eval_camera(camera, uv, sample_disk(luv));
  } else {
    const auto width  = 2.0f;
    const auto offset = 0.5f;
    auto       fuv =
        width *
            vec2f{
                puv.x < 0.5f ? sqrt(2 * puv.x) - 1 : 1 - sqrt(2 - 2 * puv.x),
                puv.y < 0.5f ? sqrt(2 * puv.y) - 1 : 1 - sqrt(2 - 2 * puv.y),
            } +
        offset;
    auto uv = vec2f{
        (ij.x + fuv.x) / image_size.x, (ij.y + fuv.y) / image_size.y};
    return eval_camera(camera, uv, sample_disk(luv));
  }
}

// Sample lights wrt solid angle
static vec3f sample_lights(const scene_data& scene, const trace_lights& lights,
    const vec3f& position, float rl, float rel, const vec2f& ruv) {
  auto  light_id = sample_uniform((int)lights.lights.size(), rl);
  auto& light    = lights.lights[light_id];
  if (light.instance != invalidid) {
    auto& instance  = scene.instances[light.instance];
    auto& shape     = scene.shapes[instance.shape];
    auto  element   = sample_discrete(light.elements_cdf, rel);
    auto  uv        = (!shape.triangles.empty()) ? sample_triangle(ruv) : ruv;
    auto  lposition = eval_position(scene, instance, element, uv);
    return normalize(lposition - position);
  } else if (light.environment != invalidid) {
    auto& environment = scene.environments[light.environment];
    if (environment.emission_tex != invalidid) {
      auto& emission_tex = scene.textures[environment.emission_tex];
      auto  idx          = sample_discrete(light.elements_cdf, rel);
      auto  uv = vec2f{((idx % emission_tex.width) + 0.5f) / emission_tex.width,
          ((idx / emission_tex.width) + 0.5f) / emission_tex.height};
      return transform_direction(environment.frame,
          {cos(uv.x * 2 * pif) * sin(uv.y * pif), cos(uv.y * pif),
              sin(uv.x * 2 * pif) * sin(uv.y * pif)});
    } else {
      return sample_sphere(ruv);
    }
  } else {
    return {0, 0, 0};
  }
}

// Sample lights pdf
static float sample_lights_pdf(const scene_data& scene, const trace_bvh& bvh,
    const trace_lights& lights, const vec3f& position, const vec3f& direction) {
  auto pdf = 0.0f;
  for (auto& light : lights.lights) {
    if (light.instance != invalidid) {
      auto& instance = scene.instances[light.instance];
      // check all intersection
      auto lpdf          = 0.0f;
      auto next_position = position;
      for (auto bounce = 0; bounce < 100; bounce++) {
        auto intersection = intersect_instance(
            bvh, scene, light.instance, {next_position, direction});
        if (!intersection.hit) break;
        // accumulate pdf
        auto lposition = eval_position(
            scene, instance, intersection.element, intersection.uv);
        auto lnormal = eval_element_normal(
            scene, instance, intersection.element);
        // prob triangle * area triangle = area triangle mesh
        auto area = light.elements_cdf.back();
        lpdf += distance_squared(lposition, position) /
                (abs(dot(lnormal, direction)) * area);
        // continue
        next_position = lposition + direction * 1e-3f;
      }
      pdf += lpdf;
    } else if (light.environment != invalidid) {
      auto& environment = scene.environments[light.environment];
      if (environment.emission_tex != invalidid) {
        auto& emission_tex = scene.textures[environment.emission_tex];
        auto  wl = transform_direction_inverse(environment.frame, direction);
        auto  texcoord = vec2f{atan2(wl.z, wl.x) / (2 * pif),
            acos(clamp(wl.y, -1.0f, 1.0f)) / pif};
        if (texcoord.x < 0) texcoord.x += 1;
        auto i = clamp(
            (int)(texcoord.x * emission_tex.width), 0, emission_tex.width - 1);
        auto j    = clamp((int)(texcoord.y * emission_tex.height), 0,
               emission_tex.height - 1);
        auto prob = sample_discrete_pdf(
                        light.elements_cdf, j * emission_tex.width + i) /
                    light.elements_cdf.back();
        auto angle = (2 * pif / emission_tex.width) *
                     (pif / emission_tex.height) *
                     sin(pif * (j + 0.5f) / emission_tex.height);
        pdf += prob / angle;
      } else {
        pdf += 1 / (4 * pif);
      }
    }
  }
  pdf *= sample_uniform_pdf((int)lights.lights.size());
  return pdf;
}

struct trace_result {
  vec3f radiance = {0, 0, 0};
  bool  hit      = false;
  vec3f albedo   = {0, 0, 0};
  vec3f normal   = {0, 0, 0};
};

// Recursive path tracing.
static trace_result trace_path(const scene_data& scene, const trace_bvh& bvh,
    const trace_lights& lights, const ray3f& ray_, rng_state& rng,
    const trace_params& params) {
  // initialize
  auto radiance      = vec3f{0, 0, 0};
  auto weight        = vec3f{1, 1, 1};
  auto ray           = ray_;
  auto volume_stack  = svector<material_point>{};
  auto max_roughness = 0.0f;
  auto hit           = false;
  auto hit_albedo    = vec3f{0, 0, 0};
  auto hit_normal    = vec3f{0, 0, 0};
  auto opbounce      = 0;

  // trace  path
  for (auto bounce = 0; bounce < params.bounces; bounce++) {
    // intersect next point
    auto intersection = intersect_scene(bvh, scene, ray);
    if (!intersection.hit) {
      if (bounce > 0 || !params.envhidden)
        radiance += weight * eval_environment(scene, ray.d);
      break;
    }

    // handle transmission if inside a volume
    auto in_volume = false;
    if (!volume_stack.empty()) {
      auto& vsdf     = volume_stack.back();
      auto  distance = sample_transmittance(
          vsdf.density, intersection.distance, rand1f(rng), rand1f(rng));
      weight *= eval_transmittance(vsdf.density, distance) /
                sample_transmittance_pdf(
                    vsdf.density, distance, intersection.distance);
      in_volume             = distance < intersection.distance;
      intersection.distance = distance;
    }

    // switch between surface and volume
    if (!in_volume) {
      // prepare shading point
      auto outgoing = -ray.d;
      auto position = eval_shading_position(scene, intersection, outgoing);
      auto normal   = eval_shading_normal(scene, intersection, outgoing);
      auto material = eval_material(scene, intersection);

      // correct roughness
      if (params.nocaustics) {
        max_roughness      = max(material.roughness, max_roughness);
        material.roughness = max_roughness;
      }

      // handle opacity
      if (material.opacity < 1 && rand1f(rng) >= material.opacity) {
        if (opbounce++ > 128) break;
        ray = {position + ray.d * 1e-2f, ray.d};
        bounce -= 1;
        continue;
      }

      // set hit variables
      if (bounce == 0) {
        hit        = true;
        hit_albedo = material.color;
        hit_normal = normal;
      }

      // accumulate emission
      radiance += weight * eval_emission(material, normal, outgoing);

      // next direction
      auto incoming = vec3f{0, 0, 0};
      if (!is_delta(material)) {
        if (rand1f(rng) < 0.5f) {
          incoming = sample_bsdfcos(
              material, normal, outgoing, rand1f(rng), rand2f(rng));
        } else {
          incoming = sample_lights(
              scene, lights, position, rand1f(rng), rand1f(rng), rand2f(rng));
        }
        if (incoming == vec3f{0, 0, 0}) break;
        weight *=
            eval_bsdfcos(material, normal, outgoing, incoming) /
            (0.5f * sample_bsdfcos_pdf(material, normal, outgoing, incoming) +
                0.5f *
                    sample_lights_pdf(scene, bvh, lights, position, incoming));
      } else {
        incoming = sample_delta(material, normal, outgoing, rand1f(rng));
        weight *= eval_delta(material, normal, outgoing, incoming) /
                  sample_delta_pdf(material, normal, outgoing, incoming);
      }

      // update volume stack
      if (is_volumetric(scene, intersection) &&
          dot(normal, outgoing) * dot(normal, incoming) < 0) {
        if (volume_stack.empty()) {
          auto material = eval_material(scene, intersection);
          volume_stack.push_back(material);
        } else {
          volume_stack.pop_back();
        }
      }

      // setup next iteration
      ray = {position, incoming};
    } else {
      // prepare shading point
      auto  outgoing = -ray.d;
      auto  position = ray.o + ray.d * intersection.distance;
      auto& vsdf     = volume_stack.back();

      // accumulate emission
      // radiance += weight * eval_volemission(emission, outgoing);

      // next direction
      auto incoming = vec3f{0, 0, 0};
      if (rand1f(rng) < 0.5f) {
        incoming = sample_scattering(vsdf, outgoing, rand1f(rng), rand2f(rng));
      } else {
        incoming = sample_lights(
            scene, lights, position, rand1f(rng), rand1f(rng), rand2f(rng));
      }
      if (incoming == vec3f{0, 0, 0}) break;
      weight *=
          eval_scattering(vsdf, outgoing, incoming) /
          (0.5f * sample_scattering_pdf(vsdf, outgoing, incoming) +
              0.5f * sample_lights_pdf(scene, bvh, lights, position, incoming));

      // setup next iteration
      ray = {position, incoming};
    }

    // check weight
    if (weight == vec3f{0, 0, 0} || !isfinite(weight)) break;

    // russian roulette
    if (bounce > 3) {
      auto rr_prob = min((float)0.99, max(weight));
      if (rand1f(rng) >= rr_prob) break;
      weight *= 1 / rr_prob;
    }
  }

  return {radiance, hit, hit_albedo, hit_normal};
}

// eval one ray, returns true if the ray terminated
static bool eval_ray(const scene_data& scene, const trace_bvh& bvh,
    const trace_lights& lights, cutrace_path& paths,
    const cutrace_intersection& intersections, int idx, rng_state& rng,
    const trace_params& params) {
  // read from globals
  // we need to write back to globals at the end if the ray is not terminated
  auto radiance      = paths.radiance[idx];
  auto weight        = paths.weights[idx];
  auto ray           = paths.rays[idx];
  auto volume_back   = paths.volume_back[idx];
  auto volume_empty  = paths.volume_empty[idx];
  auto max_roughness = paths.max_roughness[idx];
  auto opbounce      = paths.opbounces[idx];
  auto bounce        = paths.bounces[idx];

  // read intersection from globals
  auto intersection = scene_intersection{intersections.instance[idx],
      intersections.element[idx], intersections.uv[idx],
      intersections.distance[idx], intersections.hit[idx]};

  if (!intersection.hit) {
    if (bounce > 0 || !params.envhidden) {
      paths.radiance[idx] = radiance + weight * eval_environment(scene, ray.d);
    }
    return true;
  }

  // handle transmission if inside a volume
  auto in_volume = false;
  if (!volume_empty) {
    auto distance = sample_transmittance(
        volume_back.density, intersection.distance, rand1f(rng), rand1f(rng));
    weight *= eval_transmittance(volume_back.density, distance) /
              sample_transmittance_pdf(
                  volume_back.density, distance, intersection.distance);
    paths.weights[idx]    = weight;
    in_volume             = distance < intersection.distance;
    intersection.distance = distance;
  }

  // switch between surface and volume
  if (!in_volume) {
    // prepare shading point
    auto outgoing = -ray.d;
    auto position = eval_shading_position(scene, intersection, outgoing);
    auto normal   = eval_shading_normal(scene, intersection, outgoing);
    auto material = eval_material(scene, intersection);

    // correct roughness
    if (params.nocaustics) {
      max_roughness            = max(material.roughness, max_roughness);
      paths.max_roughness[idx] = max_roughness;
      material.roughness       = max_roughness;
    }

    // handle opacity
    if (material.opacity < 1 && rand1f(rng) >= material.opacity) {
      if (opbounce > 128) {
        return true;
      }
      paths.opbounces[idx] = opbounce + 1;
      paths.rays[idx]      = {position + ray.d * 1e-2f, ray.d};
      return false;
    }

    // set hit variables
    if (bounce == 0) {
      paths.hit[idx]        = true;
      paths.hit_albedo[idx] = material.color;
      paths.hit_normal[idx] = normal;
    }

    // accumulate emission
    radiance += weight * eval_emission(material, normal, outgoing);
    paths.radiance[idx] = radiance;

    // next direction
    auto incoming = vec3f{0, 0, 0};
    if (!is_delta(material)) {
      if (rand1f(rng) < 0.5f) {
        incoming = sample_bsdfcos(
            material, normal, outgoing, rand1f(rng), rand2f(rng));
      } else {
        incoming = sample_lights(
            scene, lights, position, rand1f(rng), rand1f(rng), rand2f(rng));
      }
      if (incoming == vec3f{0, 0, 0}) return true;
      weight *=
          eval_bsdfcos(material, normal, outgoing, incoming) /
          (0.5f * sample_bsdfcos_pdf(material, normal, outgoing, incoming) +
              0.5f * sample_lights_pdf(scene, bvh, lights, position, incoming));
    } else {
      incoming = sample_delta(material, normal, outgoing, rand1f(rng));
      weight *= eval_delta(material, normal, outgoing, incoming) /
                sample_delta_pdf(material, normal, outgoing, incoming);
    }

    // update volume stack
    if (is_volumetric(scene, intersection) &&
        dot(normal, outgoing) * dot(normal, incoming) < 0) {
      if (volume_empty) {
        paths.volume_back[idx] = eval_material(scene, intersection);
      } else {
        paths.volume_empty[idx] = true;
      }
    }

    // setup next iteration
    ray = {position, incoming};
  } else {
    // prepare shading point
    auto outgoing = -ray.d;
    auto position = ray.o + ray.d * intersection.distance;

    // accumulate emission
    // radiance += weight * eval_volemission(emission, outgoing);

    // next direction
    auto incoming = vec3f{0, 0, 0};
    if (rand1f(rng) < 0.5f) {
      incoming = sample_scattering(
          volume_back, outgoing, rand1f(rng), rand2f(rng));
    } else {
      incoming = sample_lights(
          scene, lights, position, rand1f(rng), rand1f(rng), rand2f(rng));
    }
    if (incoming == vec3f{0, 0, 0}) return true;
    weight *=
        eval_scattering(volume_back, outgoing, incoming) /
        (0.5f * sample_scattering_pdf(volume_back, outgoing, incoming) +
            0.5f * sample_lights_pdf(scene, bvh, lights, position, incoming));

    // setup next iteration
    ray = {position, incoming};
  }

  // check weight
  if (weight == vec3f{0, 0, 0} || !isfinite(weight)) return true;

  // russian roulette
  if (bounce > 3) {
    auto rr_prob = min((float)0.99, max(weight));
    if (rand1f(rng) >= rr_prob) return true;
    weight *= 1 / rr_prob;
  }

  // finish eval one segment
  bounce++;
  if (bounce >= params.bounces) return true;

  // write to globals
  paths.rays[idx]    = ray;
  paths.weights[idx] = weight;
  paths.bounces[idx] = bounce;

  return false;
}

// generate a new primary ray from camera, at pixel (i, j),
// and initaialize its path state
static void raygen(cutrace_state& state, const cuscene_data& scene, int i,
    int j, const trace_params& params) {
  auto& camera = scene.cameras[params.camera];
  // auto  sampler = get_trace_sampler_func(params);
  auto idx = state.width * j + i;
  auto ray = sample_camera(camera, {i, j}, {state.width, state.height},
      rand2f(state.rngs[idx]), rand2f(state.rngs[idx]), params.tentfilter);

  auto& path = state.path;

  path.radiance[idx]      = vec3f{0, 0, 0};
  path.weights[idx]       = vec3f{1, 1, 1};
  path.rays[idx]          = ray;
  path.volume_back[idx]   = {};
  path.volume_empty[idx]  = true;
  path.max_roughness[idx] = 0.0f;
  path.hit[idx]           = false;
  path.hit_albedo[idx]    = vec3f{0, 0, 0};
  path.hit_normal[idx]    = vec3f{0, 0, 0};
  path.opbounces[idx]     = 0;
  path.bounces[idx]       = 0;
}

static void trace_sample(cutrace_state& state, const cuscene_data& scene,
    const cutrace_bvh& bvh, const cutrace_lights& lights, int i, int j,
    const trace_params& params, int* num_pixels_done) {
  auto idx = state.width * j + i;

  auto result = eval_ray(scene, bvh, lights, state.path, state.intersection,
      idx, state.rngs[idx], params);

  auto sample = state.pixel_samples[idx];

  if (result) {
    // ray is terminated, update image and generate a new ray
    auto& path = state.path;

    auto radiance = path.radiance[idx];
    auto hit      = path.hit[idx];
    auto albedo   = path.hit_albedo[idx];
    auto normal   = path.hit_normal[idx];
    auto ray      = path.rays[idx];

    if (!isfinite(radiance)) radiance = {0, 0, 0};
    if (max(radiance) > params.clamp)
      radiance = radiance * (params.clamp / max(radiance));
    auto weight = 1.0f / (sample + 1);
    if (hit) {
      state.image[idx] = lerp(
          state.image[idx], {radiance.x, radiance.y, radiance.z, 1}, weight);
      state.albedo[idx] = lerp(state.albedo[idx], albedo, weight);
      state.normal[idx] = lerp(state.normal[idx], normal, weight);
    } else if (!params.envhidden && !scene.environments.empty()) {
      state.image[idx] = lerp(
          state.image[idx], {radiance.x, radiance.y, radiance.z, 1}, weight);
      state.albedo[idx] = lerp(state.albedo[idx], {1, 1, 1}, weight);
      state.normal[idx] = lerp(state.normal[idx], -ray.d, weight);
    } else {
      state.image[idx]  = lerp(state.image[idx], {0, 0, 0, 0}, weight);
      state.albedo[idx] = lerp(state.albedo[idx], {0, 0, 0}, weight);
      state.normal[idx] = lerp(state.normal[idx], -ray.d, weight);
    }

    sample++;
    state.pixel_samples[idx] = sample;

    raygen(state, scene, i, j, params);
  }

  if (sample >= state.samples + params.batch) {
    atomicAdd(num_pixels_done, 1);
  }
}

// logic phase of the wavefront algorithm
// we generate a new ray if its the first invocation, or if the
// last ray terminated due to a miss/max depth reached/opacity/rr
extern "C" __global__ void trace_pixel_logic(bool first, int* num_pixels_done) {
  // pixel index
  uint2 ij;
  ij.x = blockIdx.x * blockDim.x + threadIdx.x;
  ij.y = blockIdx.y * blockDim.y + threadIdx.y;

  if (ij.x >= globals.state.width || ij.y >= globals.state.height) {
    return;
  }

  auto idx    = ij.y * globals.state.width + ij.x;
  auto sample = globals.state.pixel_samples[idx];

  // initialize state on first sample
  if (first && globals.state.samples == 0) {
    globals.state.image[idx] = {0, 0, 0, 0};
    globals.state.rngs[idx]  = make_rng(98273987, idx * 2 + 1);
    raygen(globals.state, globals.scene, ij.x, ij.y, globals.params);
  } else {
    trace_sample(globals.state, globals.scene, globals.bvh, globals.lights,
        ij.x, ij.y, globals.params, num_pixels_done);
  }
}

// extend phase of the wavefront algorithm
// we fetch the next ray to trace from the queue, trace it,
// and write the result back to the queue
extern "C" __global__ void trace_pixel_extend() {
  // pixel index
  uint2 ij;
  ij.x = blockIdx.x * blockDim.x + threadIdx.x;
  ij.y = blockIdx.y * blockDim.y + threadIdx.y;

  if (ij.x >= globals.state.width || ij.y >= globals.state.height) {
    return;
  }

  auto idx = ij.y * globals.state.width + ij.x;
  // auto sample = globals.state.pixel_samples[idx];

  // if (sample >= globals.state.samples + globals.params.batch) {
  //   return;
  // }

  auto ray          = globals.state.path.rays[idx];
  auto intersection = intersect_scene(globals.bvh, globals.scene, ray);

  auto& intersections         = globals.state.intersection;
  intersections.instance[idx] = intersection.instance;
  intersections.element[idx]  = intersection.element;
  intersections.uv[idx]       = intersection.uv;
  intersections.distance[idx] = intersection.distance;
  intersections.hit[idx]      = intersection.hit;
}

// dispatch trace_pixel for each pixel
extern "C" void cutrace_samples(hipDeviceptr_t trace_globals) {
  auto globals_cpu = cutrace_globals{};
  auto result      = hipMemcpyDtoH(
      &globals_cpu, trace_globals, sizeof(cutrace_globals));
  if (result != hipSuccess) {
    const char* error_name;
    hipDrvGetErrorName(result, &error_name);
    printf("cutrace_samples: hipMemcpyDtoH error %s\n", error_name);
  }

  auto cpyResult = hipMemcpyToSymbol(HIP_SYMBOL(
      globals), &globals_cpu, sizeof(cutrace_globals));
  if (cpyResult != hipSuccess) {
    printf("cutrace_samples: hipMemcpyToSymbol error %s\n",
        hipGetErrorName(cpyResult));
  }

  int* num_pixels_done;
  hipMalloc(&num_pixels_done, sizeof(int));
  int num_pixels_done_cpu = 0;

  int width  = globals_cpu.state.width;
  int height = globals_cpu.state.height;

  dim3 blockSize = {16, 16, 1};
  dim3 gridSize  = {(width + blockSize.x - 1) / blockSize.x,
       (height + blockSize.y - 1) / blockSize.y, 1};

  int  cur   = 0;
  bool first = true;
  while (num_pixels_done_cpu < width * height) {
    hipMemset(num_pixels_done, 0, sizeof(int));

    trace_pixel_logic<<<gridSize, blockSize>>>(first, num_pixels_done);
    first = false;

    trace_pixel_extend<<<gridSize, blockSize>>>();

    hipMemcpy(&num_pixels_done_cpu, num_pixels_done, sizeof(int),
        hipMemcpyDeviceToHost);

    // printf("iteration %d, num pixels done: %d/%d\n", cur++,
    // num_pixels_done_cpu,
    //     width * height);
  }

  hipFree(num_pixels_done);
}

}  // namespace yocto